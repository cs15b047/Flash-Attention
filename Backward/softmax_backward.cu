#include "hip/hip_runtime.h"
#include "self_attention_backward.cuh"
#include "hipblas.h"

__global__ void elementwise_product(const float *P_, const float *dP_, float *dS_, const int N) {
    int idx1 = blockIdx.x;
    
    const float* P = P_ + idx1 * N * N, *dP = dP_ + idx1 * N * N;
    float *dS = dS_ + idx1 * N * N;

    for(int idx2 = threadIdx.x; idx2 < N * N; idx2 += blockDim.x) {
        dS[idx2] = P[idx2] * dP[idx2];
    }
    __syncthreads();
}

__global__ void subtraction(const float* P_, const float* rowsums_, float* dS_, int N) {
    int idx1 = blockIdx.x;
    const float *P = P_ + idx1 * N * N, *rowsums = rowsums_ + idx1 * N;
    float* dS = dS_ + idx1 * N * N;
    for(int idx2 = threadIdx.x; idx2 < N * N; idx2 += blockDim.x) {
        dS[idx2] = dS[idx2] - P[idx2] * rowsums[idx2 / N];
    }
    __syncthreads();
}

__global__ void rowwise_sum(const float* dS_, float* rowsums, int N) {
    int idx1 = blockIdx.x;
    const float* dS = dS_ + idx1 * N * N;
    float* rowsum = rowsums + idx1 * N;
    
    // Each thread computes sum of a row
    for (int idx2 = threadIdx.x; idx2 < N; idx2 += blockDim.x) {
        float sum = 0;
        for(int idx3 = 0; idx3 < N; idx3++) {
            sum += dS[idx2 * N + idx3];
        }
        rowsum[idx2] = sum;
    }
    __syncthreads();
}


__host__ void softmax_backward1(const float *P, const float* dP, float* dS, float* rowsums, int N, int batch_size, int num_heads) {
    int threads = 1024;
    int blocks = batch_size * num_heads;

    // dS = P .* dP
    elementwise_product<<<blocks, threads>>>(P, dP, dS, N);
    hipDeviceSynchronize();

    // rowsums = rowwise_sum(dS)
    rowwise_sum<<<blocks, threads>>>(dS, rowsums, N);
    hipDeviceSynchronize();

    // dS = dS - P .* rowsums
    subtraction<<<blocks, threads>>>(P, (const float*)rowsums, dS, N);
    hipDeviceSynchronize();
}