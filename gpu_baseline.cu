
#include <hipblas.h>
#include <stdio.h>
#include "self_attention.cuh"
#include "softmax_cublas.cuh"


void print_matrix(const float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

__host__ void self_attention(const float *Q, const float *K, const float *V, float *O, int N, int dim){
    float *intermediate, *softmax_result;
    hipMallocManaged(&intermediate, N * N  * sizeof(float));
    hipMemset(intermediate, 0, N * N  * sizeof(float));
    hipMallocManaged(&softmax_result, N * N * sizeof(float));
    hipMemset(softmax_result, 0, N * N * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                N, N, dim, &alpha, 
                K, dim, Q, dim, &beta, intermediate, N);
    hipblasDestroy(handle);
    
    softmax_cublas(intermediate, softmax_result, N);

    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                dim, N, N, &alpha, 
                V, dim, softmax_result, N, &beta, O, dim);
    hipblasDestroy(handle);
}
