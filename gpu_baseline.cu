
#include <hipblas.h>
#include <stdio.h>
#include "self_attention.cuh"


__host__ void self_attention(const float *Q, const float *K, const float *V, float *O, int N, int dim){
    float *intermediate;
    hipMallocManaged(&intermediate, N * N  * sizeof(float));
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                N, N, dim, &alpha, 
                K, N, Q, N, &beta, intermediate, N);
    hipblasDestroy(handle);

    

    
    
    

}
