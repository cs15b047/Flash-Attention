
#include <hipblas.h>
#include <stdio.h>
#include "self_attention.cuh"
#include "softmax_cublas.cuh"


void print_matrix(const float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

__host__ void self_attention(const float *Q, const float *K, const float *V, float *intermediate, float *softmax_result, float *O, int N, int dim){
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                N, N, dim, &alpha, 
                K, dim, Q, dim, &beta, intermediate, N);
    
    softmax_cublas(intermediate, softmax_result, N);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                dim, N, N, &alpha, 
                V, dim, softmax_result, N, &beta, O, dim);
    hipblasDestroy(handle);
    hipDeviceSynchronize();
}


