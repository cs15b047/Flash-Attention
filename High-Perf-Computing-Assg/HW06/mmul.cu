#include "mmul.h"
#include <stdio.h>

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n) {
    hipblasStatus_t status;
    float alpha = 1.0f;
    float beta = 0.0f;
    status = hipblasSgemm(handle, 
        HIPBLAS_OP_N, HIPBLAS_OP_N, 
        n, n, n, 
        &alpha, A, n, B, n, &beta, C, n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("!!!! kernel execution error !!!\n");
    }
}