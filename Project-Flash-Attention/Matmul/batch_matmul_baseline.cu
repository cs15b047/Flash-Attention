#include "hip/hip_runtime.h"
#include "batch_matmul.cuh"
#include <stdio.h>
using namespace std;

__global__ void matmul_tiled_coalesced_one_to_many_kernel(const float* A, const float* B, float* C, int M, int N, int K, const int BLOCK_SIZE) {
    // Block index
    int bx = blockIdx.x; //the B (and C) matrix sub-block column index
    int by = blockIdx.y; //the A (and C) matrix sub-block row index
    int bz  = blockIdx.z; 
    
    // Thread index
    int tx= threadIdx.x; //the column index in the sub-block
    int ty = threadIdx.y; //the row index in the sub-block
    // Index of the first sub-matrix of A processed by the block
    int aBegin= K * BLOCK_SIZE * by;
    // Index of the last sub-matrix of A processed by the block
    int aEnd= aBegin+ K - 1;
    // Step size used to iterate through the sub-matrices of A
    int aStep= BLOCK_SIZE;
    // Index of the first sub-matrix of B processed by the block
    int bBegin= BLOCK_SIZE * bx;
    // Step size used to iterate through the sub-matrices of B
    int bStep= BLOCK_SIZE * N;
    // The element of the block sub-matrix that is computed
    // by the thread
    float Csub0 = 0, Csub1 = 0, Csub2 = 0, Csub3 = 0; 

    extern __shared__ float shMem[];
    float* As = shMem;
    float* Bs = shMem + BLOCK_SIZE * BLOCK_SIZE;

    // Loop over all the sub-matrices (tiles) of A and B required to
    // compute the block sub-matrix; moving in A left to right in
    // a row, and in B from top to bottom in a column
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Load tiles from global memory into shared memory; each
        // thread loads one element of the two tiles from A & B
        // utilize z index to load 4 tiles
        
        As[ty * BLOCK_SIZE + tx] = A[bz*M*K + a + K* ty + tx];
        As[ty * BLOCK_SIZE + tx + BLOCK_SIZE/2] = A[bz*M*K + a + K* ty + tx + BLOCK_SIZE/2];
        As[(ty + BLOCK_SIZE/2) * BLOCK_SIZE + tx] = A[bz*M*K +a + K* (ty + BLOCK_SIZE/2) + tx];
        As[(ty + BLOCK_SIZE/2) * BLOCK_SIZE + (tx + BLOCK_SIZE/2)] = A[bz*M*K + a + K* (ty + BLOCK_SIZE/2) + tx + BLOCK_SIZE/2];

       

        
        Bs[ty * BLOCK_SIZE + tx] = B[bz*N*K + b + N* ty + tx];
        Bs[ty * BLOCK_SIZE + tx + BLOCK_SIZE/2] = B[bz*N*K + b + N* ty + tx + BLOCK_SIZE/2];
        Bs[(ty + BLOCK_SIZE/2) * BLOCK_SIZE + tx] = B[bz*N*K + b + N* (ty + BLOCK_SIZE/2) + tx];
        Bs[(ty + BLOCK_SIZE/2) * BLOCK_SIZE + (tx + BLOCK_SIZE/2)] = B[bz*N*K + b + N * (ty + BLOCK_SIZE/2) + tx + BLOCK_SIZE/2];
        // Synchronize to make sure the matrices are loaded
        __syncthreads();
        // Each thread in this block computes one element
        // of the block sub-matrix (tile). Thread with indexes
        // ty and txcomputes in this tile the entry [ty][tx].
        for (int k = 0; k < BLOCK_SIZE; ++k){
            Csub0 += As[ty * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx];
            Csub1 += As[ty * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx + BLOCK_SIZE/2];
            Csub2 += As[(ty + BLOCK_SIZE/2) * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx];
            Csub3 += As[(ty + BLOCK_SIZE/2) * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx + BLOCK_SIZE/2];
            
        }
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    // Write the block sub-matrix to global memory;
    // each thread writes one element
    int c = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[bz*N*M +c + N * ty + tx] = Csub0;
    C[bz*N*M +c + N * ty + tx + BLOCK_SIZE/2] = Csub1;
    C[bz*N*M +c + N * (ty + BLOCK_SIZE/2) + tx] = Csub2;
    C[bz*N*M +c + N * (ty + BLOCK_SIZE/2) + tx + BLOCK_SIZE/2] = Csub3;
}


__host__ void batch_matmul(const float *A, const float *B, float *C, int batchsize, int M, int N, int K) {

    dim3 grid_dim((N + (2*32 - 1)) / (2 * 32), (M + (2*32 - 1)) / (2 * 32),1);
    dim3 block_dim(32, 32);
    int shared_mem_size = 2 * 64 * 64 * sizeof(float);

    for(int i = 0; i < batchsize; i++) {
        matmul_tiled_coalesced_one_to_many_kernel<<<grid_dim, block_dim, shared_mem_size>>>(A + i * M * K, B + i * K * N, C + i * M * N, M, N, K, 64);
        hipDeviceSynchronize();
    }
   
}


__host__ void batch_matmul_single_kernel(const float *A, const float *B, float *C, int batchsize, int M, int N, int K) {
    
    dim3 grid_dim((N + (2*32 - 1)) / (2 * 32), (M + (2*32 - 1)) / (2 * 32), batchsize);
    dim3 block_dim(32, 32);
    int shared_mem_size = 2 * 64 * 64 * sizeof(float);

    matmul_tiled_coalesced_one_to_many_kernel<<<grid_dim, block_dim, shared_mem_size>>>(A, B, C, M, N, K, 64);
    hipDeviceSynchronize();   
}


